#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include <hip/hip_runtime.h>
#include <surface_functions.h>
#include <cuda_surface_types.h>
#include <>

#include "cuda_definitions.h"
#include "cuda_assert.h"

#include "Core/SceneData.cuh"
#include "BVH/MBVHNode.cuh"
#include "BVH/BVHNode.cuh"
#include "Core/Random.cuh"

using namespace glm;

#ifdef __HIPCC__
#define LAUNCH_BOUNDS __launch_bounds__(128, 8)
#else
#define LAUNCH_BOUNDS
#endif

#define USE_MICROFACETS 1
#define MAX_DEPTH 16
#define PI glm::pi<float>()
#define INVPI glm::one_over_pi<float>()

__device__ int primary_ray_cnt = 0;
//The index of the ray at which we start generating more rays in ray generation step.
//Effectively is the last index which was previously generated + 1.
__device__ int start_position = 0;
//Ray number incremented by every thread in primary_rays ray generation
__device__ int ray_nr_primary = 0;
//Ray number to fetch different ray from every CUDA thread during the extend step.
__device__ int ray_nr_extend = 0;

//Ray number to fetch different ray from every CUDA thread in the shade step.
__device__ int ray_nr_microfacet = 0;
__device__ int ray_nr_regular = 0;
__device__ int ray_nr_invalid = 0;

//Ray number to fetch different ray from every CUDA thread in the connect step.
__device__ int ray_nr_connect = 0;
//Number of shadow rays generated in shade step, which are placed in connect step.
__device__ int shadow_ray_cnt = 0;

surface<void, cudaSurfaceType2D> framebuffer;

__device__ inline void draw(unsigned int x, unsigned int y, const vec4& color)
{
	surf2Dwrite(color, framebuffer, x * sizeof(vec4), y);
}

__device__ inline void draw_unbounded(unsigned int x, unsigned int y, const vec4& color)
{
	surf2Dwrite(color, framebuffer, x * sizeof(vec4), y, hipBoundaryModeZero);
}

__global__ void setGlobals(int rayBufferSize, int width, int height)
{
	const int maxBuffer = width * height;
	const unsigned int progress = rayBufferSize - (glm::min(primary_ray_cnt, rayBufferSize));
	start_position += progress;
	start_position %= maxBuffer;

	shadow_ray_cnt = 0;
	primary_ray_cnt = 0;
	ray_nr_primary = 0;
	ray_nr_extend = 0;

	ray_nr_microfacet = 0;
	ray_nr_regular = 0;
	ray_nr_invalid = 0;

	ray_nr_connect = 0;
}

__global__ void generatePrimaryRays(
	Ray* rays,
	vec3 origin,
	vec3 viewDir,
	vec3 hor,
	vec3 ver,
	int w,
	int h,
	float invw,
	float invh,
	int rayBufferSize,
	unsigned int frame
)
{
	while (true)
	{
		const int index = atomicAdd(&ray_nr_primary, 1);

		// Start from where extended rays ended
		const int rayIdx = index + primary_ray_cnt;
		if (rayIdx >= rayBufferSize) return;

		unsigned int seed = (index + frame * 147565741) * 720898027 * index;

		const int x = (start_position + index) % w;
		const int y = ((start_position + index) / w) % h;

		const float px = float(x) + RandomFloat(seed) - 0.5f;
		const float py = float(y) + RandomFloat(seed) - 0.5f;

		rays[rayIdx] = Ray::generate(origin, viewDir, hor, ver, px, py, invw, invh, x + y * w);
	}
}

__global__ void LAUNCH_BOUNDS extend(Ray * rays, SceneData scene, int rayBufferSize)
{
	while (true)
	{
		const int index = atomicAdd(&ray_nr_extend, 1);

		if (index >= rayBufferSize) return;

		Ray & ray = rays[index];
		ray.t = MAX_DISTANCE;
		MBVHNode::traverseMBVH(ray.origin, ray.direction, &ray.t, &ray.hit_idx, scene);
	}
}

__global__ void LAUNCH_BOUNDS shade_invalid(Ray * rays, Ray * eRays, ShadowRay * sRays, SceneData scene, unsigned int frame, int rayBufferSize)
{
	while (true)
	{
		const int index = atomicAdd(&ray_nr_invalid, 1);
		if (index >= rayBufferSize) return;

		Ray & ray = rays[index];
		vec3 color = vec3(0.0f);
		float alpha = 1.0f;

		if (ray.valid())
		{
			const Material& mat = scene.gpuMaterials[scene.gpuMatIdxs[ray.hit_idx]];
			if (mat.type != Light) continue;

			ray.origin = ray.getHitpoint();
			const uvec3 tIdx = scene.indices[ray.hit_idx];
			const vec3 cN = scene.centerNormals[ray.hit_idx];
			const vec3 bary = triangle::getBaryCoords(ray.origin, cN, scene.vertices[tIdx.x], scene.vertices[tIdx.y], scene.vertices[tIdx.z]);

			const vec2 tCoords = triangle::getTexCoords(bary, scene.texCoords[tIdx.x], scene.texCoords[tIdx.y], scene.texCoords[tIdx.z]);
			vec3 normal;
			if (mat.normalTex >= 0)
			{
				vec3 T, B;
				convertToLocalSpace(cN, &T, &B);
				const vec3 n = scene.getTextureNormal(mat.normalTex, tCoords);
				normal = normalize(localToWorld(n, T, B, cN));
			}
			else
				normal = triangle::getNormal(bary, scene.normals[tIdx.x], scene.normals[tIdx.y], scene.normals[tIdx.z]);

			const bool backFacing = glm::dot(normal, ray.direction) >= 0.0f;
			if (backFacing) normal *= -1.0f;

			const vec3 matColor = mat.diffuseTex < 0 ? mat.albedo : scene.getTextureColor(mat.diffuseTex, tCoords);
			const auto mf = scene.microfacets[scene.gpuMatIdxs[ray.hit_idx]];

			if (ray.bounces <= 0)
			{
				color = mat.emission;
			}
			else if (ray.lastBounceType <= Fresnel || ray.lastBounceType >= FresnelBeckmann)
			{
				color = ray.throughput * mat.emission;
			}
			else
			{
				const float NdotL = dot(ray.lastNormal, ray.direction);
				const float LNdotL = dot(normal, -ray.direction);
				const float lightPDF = ray.t * ray.t / (LNdotL * triangle::getArea(scene.vertices[tIdx.x], scene.vertices[tIdx.y], scene.vertices[tIdx.z]));

				const vec3 wi = glm::reflect(-ray.direction, ray.lastNormal);
				float oPDF = 0;
				switch (ray.lastBounceType)
				{
				case(Lambertian):
				{
					oPDF = NdotL * glm::one_over_pi<float>();
					break;
				}
				case(GGX):
				{
					oPDF = 1.0f / mf.pdf_ggx(ray.direction, ray.lastNormal, wi);
					break;
				}
				case(Beckmann):
				{
					oPDF = 1.0f / mf.pdf_beckmann(ray.direction, ray.lastNormal, wi);
					break;
				}
				case(Trowbridge):
				{
					oPDF = 1.0f / mf.pdf_trowbridge_reitz(ray.direction, ray.lastNormal, wi);
					break;
				}
				case(FresnelGGX):
				{
					oPDF = 1.0f / mf.pdf_ggx(ray.direction, ray.lastNormal, wi);
					break;
				}
				case(FresnelBeckmann):
				{
					oPDF = 1.0f / mf.pdf_beckmann(ray.direction, ray.lastNormal, wi);
					break;
				}
				case(FresnelTrowbridge):
				{
					oPDF = 1.0f / mf.pdf_trowbridge_reitz(ray.direction, ray.lastNormal, wi);
					break;
				}
				default:
					break;
				}

				const vec3 col = ray.throughput * mat.emission * float(scene.lightCount) * NdotL;
				if (lightPDF > 0 && oPDF > 0)
				{
					const float sum = oPDF + lightPDF;
					const float w1 = oPDF / sum;
					const float w2 = lightPDF / sum;
					const float PDF = 1.0f / (w1 * oPDF + w2 * lightPDF);
					color = col * PDF;
				}
				else
				{
					alpha = 0.0f;
				}

			}
		}
		else
		{
			const vec2 uv = {
				1.0f + atan2f(ray.direction.x, -ray.direction.z) * glm::one_over_pi<float>() * 0.5f,
				1.0f - acosf(ray.direction.y) * glm::one_over_pi<float>()
			};

			color = scene.skyboxEnabled ? ray.throughput * vec3(scene.getTextureColor(scene.skyboxTexture, uv)) : vec3(0.0f);
		}

		ray.throughput = vec3(0.0f);
		atomicAdd(&scene.currentFrame[ray.index].r, color.r);
		atomicAdd(&scene.currentFrame[ray.index].g, color.g);
		atomicAdd(&scene.currentFrame[ray.index].b, color.b);
		atomicAdd(&scene.currentFrame[ray.index].a, alpha);
	}
}

__global__ void LAUNCH_BOUNDS shade_regular(Ray * rays, Ray * eRays, ShadowRay * sRays, SceneData scene, unsigned int frame, int rayBufferSize)
{
	while (true)
	{
		const int index = atomicAdd(&ray_nr_regular, 1);
		if (index >= rayBufferSize) return;

		Ray & ray = rays[index];
		if (!ray.valid()) continue;

		const Material & mat = scene.gpuMaterials[scene.gpuMatIdxs[ray.hit_idx]];
		if (mat.type == Light || mat.type >= Beckmann) continue;

		vec3 color = vec3(0.0f);
		unsigned int seed = (frame * ray.index * 147565741) * 720898027 * index;

		ray.origin = ray.getHitpoint();
		const uvec3 tIdx = scene.indices[ray.hit_idx];
		const vec3 cN = scene.centerNormals[ray.hit_idx];
		const vec3 bary = triangle::getBaryCoords(ray.origin, cN, scene.vertices[tIdx.x], scene.vertices[tIdx.y], scene.vertices[tIdx.z]);
		const vec2 tCoords = triangle::getTexCoords(bary, scene.texCoords[tIdx.x], scene.texCoords[tIdx.y], scene.texCoords[tIdx.z]);
		vec3 normal;

		if (mat.normalTex >= 0)
		{
			vec3 T, B;
			convertToLocalSpace(cN, &T, &B);
			const vec3 n = scene.getTextureNormal(mat.normalTex, tCoords);
			normal = normalize(localToWorld(n, T, B, cN));
		}
		else
		{
			normal = triangle::getNormal(bary, scene.normals[tIdx.x], scene.normals[tIdx.y], scene.normals[tIdx.z]);
		}

		const bool backFacing = glm::dot(normal, ray.direction) >= 0.0f;
		normal *= backFacing ? -1.0f : 1.0f;

		const vec3 matColor = mat.diffuseTex < 0 ? mat.albedo : scene.getTextureColor(mat.diffuseTex, tCoords);
		ray.origin += normal * EPSILON;

		switch (mat.type)
		{
		case Lambertian: {
			const int light = RandomIntMax(seed, scene.lightCount - 1);
			const uvec3 lightIdx = scene.indices[scene.lightIndices[light]];
			const vec3 lightPos = triangle::getRandomPointOnSurface(scene.vertices[lightIdx.x], scene.vertices[lightIdx.y], scene.vertices[lightIdx.z], RandomFloat(seed), RandomFloat(seed));
			vec3 L = lightPos - ray.origin;
			const float squaredDistance = dot(L, L);
			const float distance = sqrtf(squaredDistance);
			L /= distance;

			const vec3 cNormal = scene.centerNormals[scene.lightIndices[light]];
			const vec3 baryLight = triangle::getBaryCoords(lightPos, cNormal, scene.vertices[lightIdx.x], scene.vertices[lightIdx.y], scene.vertices[lightIdx.z]);
			const vec3 lightNormal = triangle::getNormal(bary, scene.normals[lightIdx.x], scene.normals[lightIdx.y], scene.normals[lightIdx.z]);

			const float NdotL = dot(normal, L);
			const float LNdotL = dot(lightNormal, -L);
			const vec3 BRDF = matColor * glm::one_over_pi<float>();

			if (NdotL > 0 && LNdotL > 0)
			{
				const float area = triangle::getArea(scene.vertices[lightIdx.x], scene.vertices[lightIdx.y], scene.vertices[lightIdx.z]);
				const float solidAngle = LNdotL * area / squaredDistance;

				const auto emission = scene.gpuMaterials[scene.gpuMatIdxs[light]].emission;
				const vec3 shadowCol = ray.throughput * BRDF * emission * NdotL * float(scene.lightCount);

				const float lambertPDF = 1.0f / NdotL * glm::one_over_pi<float>();
				const float lightPDF = 1.0f / solidAngle;

				if (lightPDF > 0 && lambertPDF > 0)
				{
					const unsigned int shadowIdx = atomicAdd(&shadow_ray_cnt, 1);

					const float sum = lambertPDF + lightPDF;
					const float w1 = lambertPDF / sum;
					const float w2 = lightPDF / sum;
					sRays[shadowIdx] = ShadowRay(
						ray.origin, std::move(L), shadowCol / (w1 * lambertPDF + w2 * lightPDF),
						distance - EPSILON, ray.index
					);
					ray.lastNormal = normal;
				}
			}

			ray.reflectCosineWeighted(RandomFloat(seed), RandomFloat(seed));
			const float NdotR = dot(normal, ray.direction);
			const float PDF = NdotR * glm::one_over_pi<float>();
			ray.lastBounceType = Lambertian;
			ray.throughput *= BRDF * NdotR / PDF;
			break;
		}
		case Specular: {
			ray.throughput *= matColor;
			ray.reflect(normal);
			ray.lastBounceType = Specular;
			break;
		}
		case Fresnel: {
			ray.throughput *= matColor;
			const vec3 dir = ray.direction;
			ray.reflect(normal);
			ray.lastBounceType = Specular;

			const float n1 = backFacing ? mat.refractIdx : 1.0f;
			const float n2 = backFacing ? 1.0f : mat.refractIdx;
			const float n = n1 / n2;
			const float cosTheta = dot(normal, -dir);
			const float k = 1.0f - (n * n) * (1.0f - cosTheta * cosTheta);

			if (k > 0)
			{
				const float a = n1 - n2;
				const float b = n1 + n2;
				const float R0 = (a * a) / (b * b);
				const float c = 1.0f - cosTheta;
				const float Fr = R0 + (1.0f - R0) * (c * c * c * c * c);

				const float r = RandomFloat(seed);
				if (r > Fr)
				{
					ray.lastBounceType = Fresnel;
					if (backFacing)
						ray.throughput *= exp(-mat.absorption * ray.t);;
					ray.origin -= EPSILON * 2.0f * normal;
					ray.direction = normalize(n * dir + normal * (n * cosTheta - sqrtf(k)));
				}
			}
			break;
		}
		default:
			break;
		}

		ray.throughput = glm::max(vec3(0.0f), ray.throughput);

		const float prob = glm::min(0.5f, glm::max(ray.throughput.x, glm::min(ray.throughput.y, ray.throughput.z)));
		if (ray.bounces < MAX_DEPTH && prob > EPSILON && prob > RandomFloat(seed))
		{
			ray.bounces++;
			ray.throughput /= prob;

			unsigned int primary_index = atomicAdd(&primary_ray_cnt, 1);
			ray.lastBounceType = mat.type;
			eRays[primary_index] = ray;
		}
		else
		{
			ray.throughput = vec3(0.0f);
			atomicAdd(&scene.currentFrame[ray.index].a, 1.0f);
		}
	}
}

__global__ void LAUNCH_BOUNDS shade_microfacet(Ray * rays, Ray * eRays, ShadowRay * sRays, SceneData scene, unsigned int frame, int rayBufferSize)
{
	while (true)
	{
		const int index = atomicAdd(&ray_nr_microfacet, 1);
		if (index >= rayBufferSize) return;

		Ray & ray = rays[index];
		if (!ray.valid()) continue;

		const Material & mat = scene.gpuMaterials[scene.gpuMatIdxs[ray.hit_idx]];
		if (mat.type == Light || mat.type < Beckmann) continue;

		vec3 color = vec3(0.0f);
		unsigned int seed = (frame * ray.index * 147565741) * 720898027 * index;

		ray.origin = ray.getHitpoint();
		const uvec3 tIdx = scene.indices[ray.hit_idx];
		const vec3 cN = scene.centerNormals[ray.hit_idx];
		const vec3 bary = triangle::getBaryCoords(ray.origin, cN, scene.vertices[tIdx.x], scene.vertices[tIdx.y], scene.vertices[tIdx.z]);
		const vec2 tCoords = triangle::getTexCoords(bary, scene.texCoords[tIdx.x], scene.texCoords[tIdx.y], scene.texCoords[tIdx.z]);
		vec3 normal;

		if (mat.normalTex >= 0)
		{
			vec3 T, B;
			convertToLocalSpace(cN, &T, &B);
			const vec3 n = scene.getTextureNormal(mat.normalTex, tCoords);
			normal = normalize(localToWorld(n, T, B, cN));
		}
		else
		{
			normal = triangle::getNormal(bary, scene.normals[tIdx.x], scene.normals[tIdx.y], scene.normals[tIdx.z]);
		}

		const bool backFacing = glm::dot(normal, ray.direction) >= 0.0f;
		normal *= backFacing ? -1.0f : 1.0f;

		const vec3 matColor = mat.diffuseTex < 0 ? mat.albedo : scene.getTextureColor(mat.diffuseTex, tCoords);
		const auto mf = scene.microfacets[scene.gpuMatIdxs[ray.hit_idx]];

		const vec3 wi = -ray.direction;

		vec3 T, B;
		convertToLocalSpace(normal, &T, &B);
		const vec3 wiLocal = normalize(vec3(dot(T, wi), dot(B, wi), dot(normal, wi)));

		vec3 wmLocal{};
		switch (mat.type)
		{
		case(Beckmann):
		{
			wmLocal = mf.sample_beckmann(wiLocal, RandomFloat(seed), RandomFloat(seed));
			break;
		}
		case(GGX):
		{
			wmLocal = mf.sample_ggx(wiLocal, RandomFloat(seed), RandomFloat(seed));
			break;
		}
		case(Trowbridge):
		{
			wmLocal = mf.sample_trowbridge_reitz(wiLocal, RandomFloat(seed), RandomFloat(seed));
			break;
		}
		case(FresnelBeckmann):
		{
			wmLocal = mf.sample_beckmann(wiLocal, RandomFloat(seed), RandomFloat(seed));

			break;
		}
		case(FresnelGGX):
		{
			wmLocal = mf.sample_ggx(wiLocal, RandomFloat(seed), RandomFloat(seed));
			break;
		}
		case(FresnelTrowbridge):
		{
			wmLocal = mf.sample_trowbridge_reitz(wiLocal, RandomFloat(seed), RandomFloat(seed));
			break;
		}
		default:
			break;
		}

		// Half-way vector
		const vec3 wm = T * wmLocal.x + B * wmLocal.y + normal * wmLocal.z;
		// Local new ray direction
		const vec3 woLocal = glm::reflect(-wiLocal, wmLocal);

		// New outgoing ray direction
		vec3 wo = localToWorld(woLocal, T, B, wm);

		float PDF = 0.0f;
		switch (mat.type)
		{
		case(Beckmann):
		{
			PDF = mf.pdf_beckmann(woLocal, wiLocal, wmLocal);
			break;
		}
		case(GGX):
		{
			PDF = mf.pdf_ggx(woLocal, wiLocal, wmLocal);
			break;
		}
		case(Trowbridge):
		{
			PDF = mf.pdf_trowbridge_reitz(woLocal, wiLocal, wmLocal);
			break;
		}
		case(FresnelBeckmann):
		{
			PDF = mf.pdf_beckmann(woLocal, wiLocal, wmLocal);

			break;
		}
		case(FresnelGGX):
		{
			PDF = mf.pdf_ggx(woLocal, wiLocal, wmLocal);
			break;
		}
		case(FresnelTrowbridge):
		{
			PDF = mf.pdf_trowbridge_reitz(woLocal, wiLocal, wmLocal);
			break;
		}
		default:
			break;
		}

		ray.origin += wm * scene.normalEpsilon;

		if (mat.type >= FresnelBeckmann)
		{
			ray.lastBounceType = mat.type;

			const float n1 = backFacing ? mat.refractIdx : 1.0f;
			const float n2 = backFacing ? 1.0f : mat.refractIdx;
			const float n = n1 / n2;
			const float cosTheta = dot(wm, wi);
			const float k = 1.0f - (n * n) * (1.0f - cosTheta * cosTheta);

			if (k > 0)
			{
				const float a = n1 - n2;
				const float b = n1 + n2;
				const float R0 = (a * a) / (b * b);
				const float c = 1.0f - cosTheta;
				const float Fr = R0 + (1.0f - R0) * (c * c * c * c * c);

				const float r = RandomFloat(seed);
				if (r > Fr)
				{
					ray.lastBounceType = Fresnel;
					if (backFacing)
						ray.throughput *= exp(-mat.absorption * ray.t);;
					ray.origin -= EPSILON * 2.0f * wm;
					wo = normalize(n * -wi + wm * (n * cosTheta - sqrtf(k)));
				}
			}
		}

		if (ray.lastBounceType != Fresnel)
		{
			const int light = RandomIntMax(seed, scene.lightCount - 1);
			const uvec3 lightIdx = scene.indices[scene.lightIndices[light]];
			const vec3 lightPos = triangle::getRandomPointOnSurface(scene.vertices[lightIdx.x], scene.vertices[lightIdx.y], scene.vertices[lightIdx.z], RandomFloat(seed), RandomFloat(seed));
			vec3 L = lightPos - ray.origin;
			const float squaredDistance = dot(L, L);
			const float distance = sqrtf(squaredDistance);
			L /= distance;

			const vec3 cNormal = scene.centerNormals[scene.lightIndices[light]];
			const vec3 baryLight = triangle::getBaryCoords(lightPos, cNormal, scene.vertices[lightIdx.x], scene.vertices[lightIdx.y], scene.vertices[lightIdx.z]);
			const vec3 lightNormal = triangle::getNormal(bary, scene.normals[lightIdx.x], scene.normals[lightIdx.y], scene.normals[lightIdx.z]);

			const float NdotL = dot(normal, L);
			const float LNdotL = dot(lightNormal, -L);

			if (NdotL > 0 && LNdotL > 0)
			{
				const float area = triangle::getArea(scene.vertices[lightIdx.x], scene.vertices[lightIdx.y], scene.vertices[lightIdx.z]);

				const auto emission = scene.gpuMaterials[scene.gpuMatIdxs[light]].emission;
				const vec3 shadowCol = ray.throughput * matColor * emission * NdotL * float(scene.lightCount);

				float mfPDF = 1.0f;
				switch (mat.type)
				{
				case(Beckmann):
				{
					mfPDF = mf.pdf_beckmann(L, wi, wm);
					break;
				}
				case(GGX):
				{
					mfPDF = mf.pdf_ggx(L, wi, wm);
					break;
				}
				case(Trowbridge):
				{
					mfPDF = mf.pdf_trowbridge_reitz(L, wi, wm);
					break;
				}
				case(FresnelBeckmann):
				{
					mfPDF = mf.pdf_beckmann(L, wi, wm);
					break;
				}
				case(FresnelGGX):
				{
					mfPDF = mf.pdf_ggx(L, wi, wm);
					break;
				}
				case(FresnelTrowbridge):
				{
					mfPDF = mf.pdf_trowbridge_reitz(L, wi, wm);
					break;
				}
				default:
					break;
				}

				mfPDF = 1.0f / mfPDF;
				const float lightPDF = squaredDistance / (LNdotL * area);

				if (lightPDF > 0 && mfPDF > 0)
				{
					const unsigned int shadowIdx = atomicAdd(&shadow_ray_cnt, 1);

					const float sum = mfPDF + lightPDF;
					const float w1 = mfPDF / sum;
					const float w2 = lightPDF / sum;
					const float pdf = 1.0f / (w1 * mfPDF + w2 * lightPDF);

					sRays[shadowIdx] = ShadowRay(
						ray.origin, L, shadowCol * pdf,
						distance - scene.distEpsilon, ray.index
					);
				}
			}
		}

		ray.throughput *= matColor * PDF;
		ray.direction = wo;

		ray.throughput = glm::max(vec3(0.0f), ray.throughput);

		const float prob = glm::min(0.5f, glm::max(ray.throughput.x, glm::min(ray.throughput.y, ray.throughput.z)));
		if (ray.bounces < MAX_DEPTH && prob > EPSILON && prob > RandomFloat(seed))
		{
			ray.bounces++;
			ray.throughput /= prob;

			unsigned int primary_index = atomicAdd(&primary_ray_cnt, 1);
			ray.lastBounceType = mat.type;
			ray.lastNormal = wm;
			eRays[primary_index] = ray;
		}
		else
		{
			ray.throughput = vec3(0.0f);
			atomicAdd(&scene.currentFrame[ray.index].a, 1.0f);
		}
	}
}

__global__ void LAUNCH_BOUNDS connect(ShadowRay * sRays, SceneData scene, int rayBufferSize)
{
	while (true)
	{
		const int index = atomicAdd(&ray_nr_connect, 1);
		if (index >= shadow_ray_cnt) return;

		const ShadowRay & ray = sRays[index];
		if (MBVHNode::traverseMBVHShadow(ray.origin, ray.direction, ray.t, scene))
		{
			atomicAdd(&scene.currentFrame[ray.index].r, ray.color.r);
			atomicAdd(&scene.currentFrame[ray.index].g, ray.color.g);
			atomicAdd(&scene.currentFrame[ray.index].b, ray.color.b);
		}
	}
}

__global__ void draw_framebuffer(vec4 * currentBuffer, int width, int height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= width || y >= height) return;

	const int index = x + y * width;
	const vec4 & color = currentBuffer[index];
	const vec3 col = vec3(color.r, color.g, color.b) / color.a;

	const vec3 exponent = vec3(1.0f / 2.2f);
	draw(x, y, vec4(glm::pow(col, exponent), 1.0f));
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t launchKernels(hipArray_const_t array, Params & params, int samples, int rayBufferSize)
{
	static int frame = 1;
	hipError_t err;

	err = cuda(BindSurfaceToArray(framebuffer, array));

	const auto* camera = params.camera;

	const vec3 w = camera->GetViewDirection();
	const vec3 up = camera->GetUp();
	const vec3 u = normalize(cross(w, up));
	const vec3 v = normalize(cross(u, w));

	vec3 hor, ver;

	if (params.width > params.height)
	{
		hor = u * camera->GetFOVDistance() * float(params.width) / float(params.height);
		ver = v * camera->GetFOVDistance();
	}
	else
	{
		hor = u * camera->GetFOVDistance();
		ver = v * camera->GetFOVDistance() * float(params.height) / float(params.width);
	}

	if (samples == 0)
		cuda(MemcpyToSymbol(primary_ray_cnt, &samples, sizeof(int)));

	generatePrimaryRays << <params.smCores * 8, 128 >> > (params.gpuRays, camera->GetPosition(), w, hor, ver, params.width, params.height,
		1.0f / float(params.width), 1.0f / float(params.height), rayBufferSize, frame);
	setGlobals << <1, 1 >> > (rayBufferSize, params.width, params.height);
	extend << <params.smCores * 8, 128 >> > (params.gpuRays, params.gpuScene, rayBufferSize);

	shade_regular << <params.smCores * 8, 128 >> > (params.gpuRays, params.gpuNextRays, params.gpuShadowRays, params.gpuScene, frame, rayBufferSize);
	shade_invalid << <params.smCores * 8, 128 >> > (params.gpuRays, params.gpuNextRays, params.gpuShadowRays, params.gpuScene, frame, rayBufferSize);
	shade_microfacet << <params.smCores * 8, 128 >> > (params.gpuRays, params.gpuNextRays, params.gpuShadowRays, params.gpuScene, frame, rayBufferSize);

	connect << <params.smCores * 8, 128 >> > (params.gpuShadowRays, params.gpuScene, rayBufferSize);

	dim3 dimBlock(16, 16);
	dim3 dimGrid((params.width + dimBlock.x - 1) / dimBlock.x, (params.height + dimBlock.y - 1) / dimBlock.y);
	draw_framebuffer << <dimGrid, dimBlock >> > (params.gpuScene.currentFrame, params.width, params.height);

	cuda(DeviceSynchronize());

	frame++;
	if (frame == INT_MAX) frame = 1;
	std::swap(params.gpuRays, params.gpuNextRays);
	return err;
}